#include "hip/hip_runtime.h"
/* NiuTrans.Tensor - an open-source tensor library
 * Copyright (C) 2017, Natural Language Processing Lab, Northestern University.
 * All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
 * $Created by: Xu Chen (email: hello_master1954@163.com) 2018-11-30
 */

#include "CopyIndexed.cuh"
#include "../../XDevice.h"
#include "../../XUtility.h"

namespace nts { // namespace nts(NiuTrans.Tensor)

#ifdef USE_CUDA

/*
copy selected sub-tensors where indeces are kept in tensors (kenerl version)

>> s - the source tensor
>> t - the target tensor
>> dim - the leading dimension to define "sub-tensors"
         e.g., for a tensor of size (3, 2, 4) and dim = 2, 
         we have 4 sub-tensors of size (3, 2)
>> srcIndex - the tensor to save the index of the source sub-tensors
>> tgtIndex - the tensor to save the index of the target sub-tensors
>> copyNum - number of the sub-tensors we copy for each source index, 
             e.g., for srcIndex = [1,4] and copyNum = 2,
             we actually copy the source sub-tensors 1, 2, 4, 5
*/
__global__
void KernelCopyIndexed(DTYPE * sData, DTYPE * tData, int * sIndex, int * tIndex, 
                       int blockNum, int blockSizeSrc, int blockSizeTgt, 
                       int stride, int indexSize, int copyNum)
{
    __shared__ DTYPE * sp[MAX_CUDA_THREAD_NUM_PER_BLOCK];
    __shared__ DTYPE * tp[MAX_CUDA_THREAD_NUM_PER_BLOCK];

    /* block id */
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    /* offset in each block */
    int offset = blockDim.y * blockIdx.y + threadIdx.y;

    if(i >= blockNum * indexSize * copyNum || offset >= stride)
        return;

    int realIndexSize = indexSize * copyNum;

    int realBlockNum = i / realIndexSize;
    int realIndex = i % realIndexSize;

    int realSrcIndex = sIndex[realIndex / copyNum] + realIndex % copyNum;
    int realTgtIndex = tIndex[realIndex / copyNum] + realIndex % copyNum;

    if(threadIdx.y == 0){
        sp[threadIdx.x] = sData + realBlockNum * blockSizeSrc + realSrcIndex * stride;
        tp[threadIdx.x] = tData + realBlockNum * blockSizeTgt + realTgtIndex * stride;
    }

    __syncthreads();

    DTYPE * s = sp[threadIdx.x];
    DTYPE * t = tp[threadIdx.x];

    t[offset] = s[offset];
}

/*
copy selected sub-tensors where indeces are kept in tensors

>> s - the source tensor
>> t - the target tensor
>> dim - the leading dimension to define "sub-tensors"
         e.g., for a tensor of size (3, 2, 4) and dim = 2, 
         we have 4 sub-tensors of size (3, 2)
>> srcIndex - the tensor to save the index of the source sub-tensors
>> tgtIndex - the tensor to save the index of the target sub-tensors
>> copyNum - number of the sub-tensors we copy for each source index, 
             e.g., for srcIndex = [1,4] and copyNum = 2,
             we actually copy the source sub-tensors 1, 2, 4, 5
*/
void _CudaCopyIndexed(const XTensor * s, XTensor * t, int dim,
                      const XTensor * srcIndex, const XTensor * tgtIndex,
                      int copyNum)
{
    int devID = s->devID;
    int order = s->order;
    int indexSize = srcIndex->unitNum;

    int blockNum = 1;
    int stride = 1;
    int blockSizeSrc = 1;
    int blockSizeTgt = 1;

    for (int i = 0; i < dim; i++)
        blockNum *= s->GetDim(i);
    
    for (int i = dim + 1; i < order; i++)
        stride *= s->GetDim(i);

    blockSizeSrc = stride * s->GetDim(dim);
    blockSizeTgt = stride * t->GetDim(dim);

    int cudaGrids[3];
    int cudaBlocks[3];

    int devIDBackup;
    ProtectCudaDev(devID, devIDBackup);

    GDevs.GetCudaThread2D(devID, blockNum * indexSize * copyNum, stride, MAX_INT, cudaGrids, cudaBlocks);

    dim3 blocks(cudaGrids[0], cudaGrids[1]);
    dim3 threads(cudaBlocks[0], cudaBlocks[1]);

    DTYPE * sData = (DTYPE*)s->data;
    DTYPE * tData = (DTYPE*)t->data;

    int * sIndex = (int *)srcIndex->data;
    int * tIndex = (int *)tgtIndex->data;

    KernelCopyIndexed<<<blocks, threads >>>(sData, tData, sIndex, tIndex, 
                                            blockNum, blockSizeSrc, blockSizeTgt,
                                            stride, indexSize, copyNum);

    BacktoCudaDev(devID, devIDBackup);

}

#endif // USE_CUDA

} // namespace nts(NiuTrans.Tensor)