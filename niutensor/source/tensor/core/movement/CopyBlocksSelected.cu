#include "hip/hip_runtime.h"
/* NiuTrans.Tensor - an open-source tensor library
* Copyright (C) 2017, Natural Language Processing Lab, Northestern University.
* All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*   http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

/*
* $Created by: XIAO Tong (email: xiaotong@mail.neu.edu.cn) 2018-04-24
*/

#include "CopyBlocks.h"
#include "CopyBlocksSelected.cuh"
#include "../../XUtility.h"
#include "../../XDevice.h"

namespace nts { // namespace nts(NiuTrans.Tensor)

#ifdef USE_CUDA

/*
copy a number of blocks from source positions to target positions
>> source - data array (head of the blocks) to copy from
>> blockSize - size of block
>> sourceBlocks - source positions of the copy
>> blockNum - number of blocks
>> target - target data array
>> targetBlocks - target positions of the copy
*/
__global__
void KernelCopyBlocksSelected(DTYPE * source, int blockSize, int * sourceBlocks, int blockNum, DTYPE * target, int * targetBlocks)
{
    /* block index */
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    /* entry index in the block */
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if (j >= blockNum)
        return;

    /* target position */
    int srcIndex = sourceBlocks[j];
    int tgtIndex = targetBlocks[j];

    DTYPE * s = source + blockSize * srcIndex;
    DTYPE * t = target + blockSize * tgtIndex;

    if (i < blockSize)
        t[i] = s[i];
}

/*
copy a number of blocks from source positions to target positions (cuda version)
>> source - data array (head of the blocks) to copy from
>> blockSize - size of block
>> sourceBlocks - source positions of the copy
>> blockNum - number of blocks
>> target - target data array
>> targetBlocks - target positions of the copy
>> myMem - memory pool
*/
void _CudaCopyBlocksSelected(void * source, int blockSize, int * sourceBlocks, int blockNum, void * target, int * targetBlocks, XMem * myMem, int devID)
{
    CheckNTErrors(devID >= 0, "Wrong device to run!");
    CheckNTErrors((blockSize % sizeof(DTYPE) == 0), "Unsupported block size!");

    int devIDBackup;
    ProtectCudaDev(devID, devIDBackup);

    /* copy the index to the GPU memory */
    int * sourceBlocksTMP = myMem != NULL ? 
                           (int*)myMem->AllocBuf(myMem->devID, blockNum * sizeof(int)) : 
                           (int *)XMemAlloc(devID, blockNum * sizeof(int));
    int * targetBlocksTMP = myMem != NULL ? 
                           (int*)myMem->AllocBuf(myMem->devID, blockNum * sizeof(int)) : 
                           (int *)XMemAlloc(devID, blockNum * sizeof(int));
    
    XMemCopy(sourceBlocksTMP, devID, sourceBlocks, -1, blockNum * sizeof(int));
    XMemCopy(targetBlocksTMP, devID, targetBlocks, -1, blockNum * sizeof(int));

    int cudaGrids[3];
    int cudaBlocks[3];

    GDevs.GetCudaThread2D(devID, blockSize / sizeof(DTYPE), blockNum, MAX_INT, cudaGrids, cudaBlocks);

    KernelCopyBlocksSelected << <dim3(cudaGrids[0], cudaGrids[1]), dim3(cudaBlocks[0], cudaBlocks[1]) >> >
                               ((DTYPE*)source, blockSize / sizeof(DTYPE), sourceBlocksTMP, blockNum, (DTYPE*)target, targetBlocksTMP);
    
    if (myMem != NULL) {
        myMem->ReleaseBuf(myMem->devID, blockNum * sizeof(int));
        myMem->ReleaseBuf(myMem->devID, blockNum * sizeof(int));
    }
    else {
        XMemFree(devID, sourceBlocksTMP);
        XMemFree(devID, targetBlocksTMP);
    }

    BacktoCudaDev(devID, devIDBackup);
}

#endif // USE_CUDA

} // namespace nts(NiuTrans.Tensor)