#include "hip/hip_runtime.h"
/* NiuTrans.Tensor - an open-source tensor library
 * Copyright (C) 2017, Natural Language Processing Lab, Northestern University. 
 * All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
* $Created by: XIAO Tong (email: xiaotong@mail.neu.edu.cn) 2018-04-25
*/

#include "HardTanH.h"
#include "HardTanH.cuh"
#include "../XDevice.h"

namespace nts{ // namespace nts(NiuTrans.Tensor)

#ifdef USE_CUDA

/* 
hard tanh forward computation (Cuda kernel) 
y =  1    if x > 1
     x    if -1 <= x <= 1
    -1    if x < -1
>> x - input data array
>> y - output data array
>> size - size of input/output
*/
__global__ 
void KernelHardtanhCompute(DTYPE * x, DTYPE * y, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size){
        DTYPE p = x[i];
        if(p > (DTYPE)1.0)
            p = (DTYPE)1.0;
        else if(p < (DTYPE)-1.0)
            p = (DTYPE)-1.0;
        y[i] = p;
    }
}

/*
hard tanh forward computation (Cuda version) 
y =  1    if x > 1
     x    if -1 <= x <= 1
    -1    if x < -1
>> x - input tensor
>> y - output tensor
*/
void _CudaHardTanH(const XTensor * x, XTensor * y)
{
    CheckNTErrors(!x->isSparse && !y->isSparse, 
                  "The hard tanh activation function does not support sparse tensors.");

    int gridSize[3], blockSize[3];

    GDevs.GetCudaThread(x->devID, x->unitNum, gridSize, blockSize);

    int devIDBackup;
    ProtectCudaDev(x->devID, devIDBackup);

    KernelHardtanhCompute<<<dim3(gridSize[0]), dim3(blockSize[0])>>>((DTYPE*)x->data, (DTYPE*)y->data, x->unitNum);

    BacktoCudaDev(x->devID, devIDBackup);
}

/* 
hard tanh backward computation of dE/dx (Cuda kernel)

dy/dx = 1     if -1 <= x <= 1
        0     otherwise

>> dedy - dE/dy
>> dedx - dE/dx
>> y - y of the function
>> x - x of the function
>> size - size of y/x
*/
__global__ 
void KernelHardtanhBackward(DTYPE * dedy, DTYPE * dedx, DTYPE * x, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size){
        DTYPE s = x[i];
        if(s > (DTYPE)1.0 || s < (DTYPE)-1.0)
            dedx[i] = 0;
        else
            dedx[i] = dedy[i];
    }
}

/*
backward computation (Cuda version)

dE/dx = dE/dy * dy/dx

hard tanh: y =  1    if x > 1
                x    if -1 <= x <= 1
               -1    if x< -1

   and dy/dx =  1    if -1 <= x <= 1
                0    otherwise

>> y - output of the hardtanh function
>> x - input of the hardtanh function
>> dedy - dE/dy
>> dedx - dE/dx
*/
void _CudaHardTanHBackward(XTensor * y, XTensor * x, 
                           XTensor * dedy, XTensor * dedx)
{
    int gridSize[3], blockSize[3];

    GDevs.GetCudaThread(x->devID, x->unitNum, gridSize, blockSize);

    int devIDBackup;
    ProtectCudaDev(x->devID, devIDBackup);

    /* dE/dx = dE/dy * dy/dx */
    KernelHardtanhBackward<<<dim3(gridSize[0]),dim3(blockSize[0])>>>
                            ((DTYPE*)dedy->data, 
                            (DTYPE*)dedx->data,
                            (DTYPE*)x->data, 
                             x->unitNum);

    BacktoCudaDev(x->devID, devIDBackup);
}

#endif

} // namespace nts(NiuTrans.Tensor)